#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
void parallelor::copydata(int s,vector<edge>&edges,int nodenum){
	
};
void parallelor::dellocate(){
};
void parallelor::allocate(int maxn,int maxedge){
}
void parallelor::topsort()
{
	cout<<" in top sort "<<endl;
	queue<int>zero;
	vector<int>order(nodenum*LY,-1);
	for(int i=0;i<nodenum*LY;i++)
		zero.push(i);
	int biao=0;
	while(!zero.empty())
	{
		int node=zero.front();
		zero.pop();
		order[node]=biao++;
		for(int i=0;i<neibn[node].size();i++)
		{
			if((--ancestor[neibn[node][i]])==0)
				zero.push(neibn[node][i]);
		}
	}
	vector<pair<int,int>>tmp;
	for(int i=0;i<order.size();i++)
		tmp.push_back(make_pair(i,order[i]));
	//sort(tmp.begin(),tmp.end(),pairless());
	for(int i=0;i<order.size();i++)
		ordernode.push_back(tmp[i].first);
};
void parallelor::init(pair<vector<edge>,vector<vector<int>>>ext,vector<pair<int,int>>stpair,vector<vector<int>>&relate,ginfo ginf)
{
	//cout<<"in cuda init"<<endl;
	nodenum=ginf.pnodesize;
	edges=ext.first;
	vector<vector<int>>esigns;
	esigns=ext.second;
	stp=stpair;
	mark=new int;
	*mark=0;
	W=WD+1;
	st=new int[2*edges.size()*LY];
	te=new int[2*edges.size()*LY];
	d=new int[nodenum*LY*YE];
	esignes=new int[edges.size()*LY];
	vector<vector<int>>nein(nodenum*LY,vector<int>());
	neibn=nein;
	vector<vector<int>>neie(nodenum,vector<int>());
	for(int i=0;i<edges.size();i++)
		{
			int s=edges[i].s;
			int t=edges[i].t;
			neibn[s].push_back(t);
			neibn[t].push_back(s);
			neie[s].push_back(i);
			neie[t].push_back(i);
		}
	int count=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<nodenum;i++)
			for(int j=0;j<neibn[i].size();j++)
			{
				st[count]=i;
				if(esigns[k][neie[i][j]]==-1)
					te[count]=i;
				else
					te[count]=neibn[i][j];
				count++;
			}
	for(int i=0;i<nodenum*LY*YE;i++)
		d[i]=WD+1;
	for(int k=0;k<LY;k++)
	{
		int boff=k*YE*nodenum;
		for(int i=0;i<YE;i++)
		{
			int soff=i*nodenum;
			for(int j=0;j<stpair.size();j++)
				d[boff+soff+stpair[i].first]=0;
		}
	}
	hipMalloc((void**)&dev_st,2*LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_te,2*LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_d,YE*LY*nodenum*sizeof(int));
	if(dev_d==NULL) {
		printf("couldn't allocate %d int's.\n");
	}
	//hipMemGetInfo()
	hipMemcpy(dev_te,te,2*LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_st,st,2*LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_d,d,YE*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	cout<<nodenum<<endl;
};
parallelor::parallelor()
{
};

__global__ void BFSfast(int *st,int *te,int *d,int round,int E,int N,int size)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;
	int eid=(i%(E*LY));
	int s=st[eid],t=te[eid];
	int ye=i/(E*LY);
	int ly=eid/E;
	int off=ye*N+ly*N*YE;
	//if(d[s+off]==round-1&&d[t+off]>round)
	d[t+off]=1;//round;
}
vector<vector<int>> parallelor::routalg(int s,int t,int bw)
{
	cout<<"blasting "<<endl;
	int kk=1;
	time_t start,end;
	start=clock();
	int size=2*edges.size()*LY*YE;
	for(int i=1;i<=WD;i++)
		BFSfast<<<size/512+1,512>>>(dev_st,dev_te,dev_d,i,2*edges.size(),nodenum,size);
	hipMemcpy(d,dev_d,nodenum*LY*YE*sizeof(int),hipMemcpyDeviceToHost);
	hipStreamSynchronize(0);
	end=clock();
	cout<<"GPU time is : "<<end-start<<endl;
	cout<<"over!"<<endl;
	vector<vector<int>>result(LY,vector<int>());
	for(int k=0;k<LY;k++)
	{
		int woff=k*YE*nodenum;
		for(int i=0;i<YE;i++)
		{
			result[k].push_back(d[woff+i*nodenum+stp[i].second]);
		}
	}
	hipFree(dev_te);
	hipFree(dev_st);
	hipFree(dev_d);
	cout<<"before return"<<endl;
	return result;
};
int fls(int x)
{
	int position;
	int i;
	if(x!=0)
		for(i=(x>>1),position=0;i!=0;++position)
			i>>=1;
	else
		position=-1;
	return pow(2,position+1);
}